// RUN: dpct --format-range=none -out-root %T/cudaMallocHost-migration-optimization %s --optimize-migration --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cudaMallocHost-migration-optimization/cudaMallocHost-migration-optimization.dp.cpp
#include <hip/hip_runtime.h>
#include <stdio.h>

void a(float *p){};
void b(float p){};
void test1(){
    float *p;
// CHECK: p = (float *)malloc(10);
    hipHostMalloc(&p, 10, hipHostMallocDefault);
// CHECK: free(p);
    hipHostFree(p);
}

void test2(){
    float *p;
// CHECK: p = (float *)sycl::malloc_host(10, q_ct1);
    hipHostMalloc(&p, 10, hipHostMallocDefault);
    a(p);
// CHECK: sycl::free(p, q_ct1);
    hipHostFree(p);
}

void test3(){
    float *p;
// CHECK: p = (float *)malloc(10);
    hipHostMalloc(&p, 10, hipHostMallocDefault);
    printf("%f", p);
// CHECK: free(p);
    hipHostFree(p);
}

void test4(){
    float *p;
// CHECK: p = (float *)malloc(10);
    hipHostMalloc(&p, 10, hipHostMallocDefault);
    b(*p);
    b(p[0]);
// CHECK: free(p);
    hipHostFree(p);
}

int main(){
  test1();
  test2();
  test3();
  return 0;
}