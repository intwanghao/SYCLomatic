// RUN: dpct --format-range=none --use-experimental-features=virtual_memory -out-root %T/virtual_memory %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/virtual_memory/virtual_memory.dp.cpp
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 100

int main() {
    hipInit(0);
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);

// CHECK:    dpct::experimental::mem_prop prop = {};
// CHECK:    prop.type = dpct::experimental::mem_allocation_type::mem_allocation_type_default;
// CHECK:    prop.location.type = dpct::experimental::mem_location_type::mem_location_type_device;
// CHECK:    prop.location.id = device;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;
    size_t granularity;
// CHECK:    dpct::experimental::mem_get_allocation_granularity(&granularity, &prop, dpct::experimental::granularity_flags::granularity_flags_minimum);
    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
    size_t POOL_SIZE =  granularity;

// CHECK:    dpct::device_ptr reserved_addr;
// CHECK:    dpct::experimental::mem_handle allocHandle;
// CHECK:    dpct::experimental::mem_address_reserve(&reserved_addr, POOL_SIZE, 0, 0, 0);
// CHECK:    dpct::experimental::mem_create(&allocHandle, POOL_SIZE, &prop, 0);
// CHECK:    dpct::experimental::mem_map(reserved_addr, POOL_SIZE, 0, allocHandle, 0);
    hipDeviceptr_t reserved_addr;
    hipMemGenericAllocationHandle_t allocHandle;
    hipMemAddressReserve(&reserved_addr, POOL_SIZE, 0, 0, 0);
    hipMemCreate(&allocHandle, POOL_SIZE, &prop, 0);
    hipMemMap(reserved_addr, POOL_SIZE, 0, allocHandle, 0);

// CHECK:    dpct::experimental::mem_access_desc accessDesc = {};
// CHECK:    accessDesc.location.type = dpct::experimental::mem_location_type::mem_location_type_device;
// CHECK:    accessDesc.location.id = device;
// CHECK:    accessDesc.flags = dpct::experimental::address_access_flags::address_access_flags_read_write;
// CHECK:    dpct::experimental::mem_set_access(reserved_addr, POOL_SIZE, &accessDesc, 1);
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    hipMemSetAccess(reserved_addr, POOL_SIZE, &accessDesc, 1);
    int* host_data = new int[SIZE];
    int* host_data2 = new int[SIZE];
    for (int i = 0; i < SIZE; ++i) {
        host_data[i] = i;
        host_data2[i] = 0;
    }

    hipMemcpyHtoD(reserved_addr, host_data, SIZE * sizeof(int));
    hipMemcpyDtoH(host_data2, reserved_addr, SIZE * sizeof(int));

    for (int i = 0; i < SIZE; ++i) {
        if(host_data[i] != host_data2[i]) {
          std::cout << "test failed" << std::endl;
          exit(-1);
        }
    }
    std::cout << "test passed" << std::endl;

// CHECK:    dpct::experimental::mem_unmap(reserved_addr, POOL_SIZE);
// CHECK:    dpct::experimental::mem_release(allocHandle);
// CHECK:    dpct::experimental::mem_address_free(reserved_addr, POOL_SIZE);
    hipMemUnmap(reserved_addr, POOL_SIZE);
    hipMemRelease(allocHandle);
    hipMemAddressFree(reserved_addr, POOL_SIZE);

    delete[] host_data;
    delete[] host_data2;

    hipCtxDestroy(context);
    return 0;
}
