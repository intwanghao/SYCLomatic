// RUN: dpct --format-range=none --optimize-migration -out-root %T/memcpy_optimization %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memcpy_optimization/memcpy_optimization.dp.cpp %s
#include <hip/hip_runtime.h>
#include <iostream>
int main(){

    float *a, *b, *c, *d, *e, *f;
    bool sync;
    c = (float *)malloc(10);
    d = (float *)malloc(10);
//CHECK:  /*
//CHECK:  DPCT1114:{{[0-9]+}}: cudaMemcpy is migrated to asynchronization memcpy, assuming in the original code the source host memory is pageable memory. If the memory is not pageable, call wait() on event return by memcpy API to ensure synchronization behavior.
//CHECK:  */
//CHECK:  q_ct1.memcpy(a, b, 10);
//CHECK:  q_ct1.memcpy(a, b, 10);
    hipMemcpy(a, b, 10, hipMemcpyHostToDevice);
    hipMemcpy(a, b, 10, hipMemcpyDeviceToDevice);

//CHECK:  for(int i = 0; i < 10; i++) {
//CHECK:    int src;
//CHECK:    q_ct1.memcpy(a, &src, 10).wait();
//CHECK:  }
    for(int i = 0; i < 10; i++) {
      int src;
      hipMemcpy(a, &src, 10, hipMemcpyHostToDevice);
    }

//CHECK:  q_ct1.memcpy(a, c, 10).wait();
//CHECK:  free(c);
    hipMemcpy(a, c, 10, hipMemcpyHostToDevice);
    free(c);

//CHECK:  q_ct1.memcpy(a, d, 10);
//CHECK:  dev_ct1.queues_wait_and_throw();
//CHECK:  free(d);
    hipMemcpy(a, d, 10, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    free(d);

//CHECK:  if(DPCT_CHECK_ERROR(q_ct1.memcpy(a, e, 10))) {
//CHECK:    std::cout << "failed" << std::endl;
//CHECK:  }
    if(hipMemcpy(a, e, 10, hipMemcpyHostToDevice)) {
        std::cout << "failed" << std::endl;
    }

//CHECK:  q_ct1.memcpy(a, f, 10).wait();
//CHECK:  if(sync) {
//CHECK:    dev_ct1.queues_wait_and_throw();
//CHECK:  }
    free(f);
    hipMemcpy(a, f, 10, hipMemcpyHostToDevice);
    if(sync) {
      hipDeviceSynchronize();
    }
    free(f);

    return 0;
}