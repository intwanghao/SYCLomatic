// RUN: dpct --format-range=none --enable-default-queue-synchronization -out-root %T/default_stream_sync %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/default_stream_sync/default_stream_sync.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/default_stream_sync/default_stream_sync.dp.cpp -o %T/default_stream_sync/default_stream_sync.dp.o %}
#include<hip/hip_runtime.h>

__global__ void kernel(int *a){

}

int main() {
  int *a, *b;
  hipStream_t s1;
  hipStreamCreate(&s1);
  hipMallocManaged(&a, 100);
  hipMallocManaged(&b, 100);

  hipMemcpyAsync(a,b, 1, hipMemcpyHostToDevice, s1);
//CHECK:  q_ct1.submit(
//CHECK:      [&](sycl::handler &cgh) {
//CHECK:        dpct::get_current_device().synchronize_with_none_default_queue();
//CHECK:        cgh.parallel_for(
//CHECK:          sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
//CHECK:          [=](sycl::nd_item<3> item_ct1) {
//CHECK:            kernel(a);
//CHECK:          });
//CHECK:      });
  kernel<<<1,1>>>(a);
  hipMemcpyAsync(b, a, 1, hipMemcpyDeviceToHost, s1);
  hipDeviceSynchronize();

  return 0;
}
