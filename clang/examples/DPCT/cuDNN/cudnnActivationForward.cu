
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(cudnnHandle_t h, cudnnActivationDescriptor_t desc, void *alpha,
          void *beta, cudnnTensorDescriptor_t dst_d, void *dst,
          cudnnTensorDescriptor_t diff_dst_d, void *diff_dst,
          cudnnTensorDescriptor_t diff_src_d, void *diff_src,
          cudnnTensorDescriptor_t src_d, void *src) {
  // Start
  cudnnActivationForward(
      h /*cudnnHandle_t*/, desc /*cudnnActivationDescriptor_t*/,
      alpha /*void **/, src_d /*cudnnTensorDescriptor_t*/, src /*void **/,
      beta /*void **/, dst_d /*cudnnTensorDescriptor_t*/, dst /*void **/);
  // End
}
