
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(cudnnHandle_t h, cudnnActivationDescriptor_t desc, void *alpha,
          void *beta, cudnnTensorDescriptor_t dst_d, void *dst,
          cudnnTensorDescriptor_t diff_dst_d, void *diff_dst,
          cudnnTensorDescriptor_t diff_src_d, void *diff_src,
          cudnnTensorDescriptor_t src_d, void *src) {
  // Start
  cudnnActivationBackward(
      h /*cudnnHandle_t*/, desc /*cudnnActivationDescriptor_t*/,
      alpha /*void **/, dst_d /*cudnnTensorDescriptor_t*/, dst /*void **/,
      diff_dst_d /*cudnnTensorDescriptor_t*/, diff_dst /*void **/,
      src_d /*cudnnTensorDescriptor_t*/, src /*void **/, beta /*void **/,
      diff_src_d /*cudnnTensorDescriptor_t*/, diff_src /*void **/);
  // End
}
